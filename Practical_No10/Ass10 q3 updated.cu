#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
 
using std::accumulate;
using std::generate;
using std::cout;
using std::vector;
#define SHMEM_SIZE 16

void init(vector<int>& h_v){
    for(int i=0;i<h_v.size();++i){
        h_v[i]=rand()%10;
    }
}

__global__ void prefixSum(int *v, int *v_r) {
    // Allocate shared memory
    __shared__ int partial_sum[SHMEM_SIZE];
 
    // Calculate thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
    // Load elements into shared memory
    partial_sum[threadIdx.x] = v[tid];
    __syncthreads();
 
    // Iterate of log base 2 the block dimension
    for (int s = 1; s < blockDim.x; s *= 2) {
        // Reduce the threads performing work by half previous the previous
        // iteration each cycle
        if (threadIdx.x % (2 * s) == 0) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }
 
    // Let the thread 0 for this block write it's result to main memory
    // Result is inexed by this block
    if (threadIdx.x == 0) {
        v_r[blockIdx.x] = partial_sum[0];
    }
}

 __global__ void scan(int *g_odata, int *g_idata, int n) 
 {   
    extern __shared__ int temp[16]; // allocated on invocation 
    
    int thid = threadIdx.x;   
    
    
    
    temp[thid] =g_idata[thid];  
    
    __syncthreads();   
    //cout<<thid<<" "<<temp[pout*n + thid]<<"\n";
    for (int offset = 1; offset < n; offset *= 2)   
    {     
        
        if (thid >= offset)       
            temp[thid] = temp[thid - offset]+ temp[thid];     
            
        __syncthreads();   
    }   
    g_odata[thid] = temp[thid]; 
    // write output 
} 
 
int main() {
    // Vector size
    int N = 1 << 16;
    N = 16;
    size_t bytes = N * sizeof(int);
 
    // Host data
    vector<int> h_v(N);
    vector<int> h_v_r(N+1);
 
  // Initialize the input data
  
      init(h_v);
     for(int i=0;i<N;++i){
        cout<<h_v[i]<<" ";
    }
    cout<<"\n";
    // Allocate device memory
    int *d_v, *d_v_r;
    hipMallocManaged(&d_v, bytes);
    hipMallocManaged(&d_v_r, bytes);
    
    // Copy to device
    hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);
    
    for(int i=0;i<N;++i){
        cout<<d_v[i]<<" ";
    }
    // TB Size
    cout<<"\n";
    const int TB_SIZE = 16;
 
    // Grid Size (No padding)
    int GRID_SIZE = 1;
 
    // Call kernels
    //prefixSum<<<1, N>>>(d_v, d_v_r);
    scan<<<1,N>>>(d_v_r,d_v,N);
    //prefixSum<<<1, TB_SIZE>>> (d_v_r, d_v_r);
 
    // Copy to host;
    hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);
 
    // Print the result
    //assert(h_v_r[0] == std::accumulate(h_v.begin(), h_v.end(), 0));
     
     
      for(int i=0;i<N;++i){
        cout<<h_v_r[i]<<" ";
    }
     
    cout << "COMPLETED SUCCESSFULLY\n";
 
    return 0;
}
