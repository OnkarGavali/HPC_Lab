#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
 
using std::accumulate;
using std::generate;
using std::cout;
using std::vector;
#define SHMEM_SIZE 1024

void init(vector<int>& h_v){
    for(int i=0;i<h_v.size();++i){
        h_v[i]=rand()%10;
    }
}

 __global__ void scan(int *g_odata, int *g_idata, int *tmp, int n ) 
 {   
    extern __shared__ int temp[1024];
    int thid = threadIdx.x;
    int dataid = blockDim.x*blockIdx.x+threadIdx.x;
    temp[thid] =g_idata[dataid];  
    __syncthreads();   
    for (int offset = 1; offset < n && offset<blockDim.x; offset *= 2)   
    {     
        if (thid >= offset)       
            temp[thid] = temp[thid - offset]+ temp[thid];     
        __syncthreads();   
    }   
    g_odata[dataid] = temp[thid];
    if(thid==blockDim.x-1){tmp[blockIdx.x]=temp[thid];}
} 
 
 /*__global__ void block_scan(int *g_odata,int *tmp, int n) 
 {   
    int dataid = blockDim.x*blockIdx.x+threadIdx.x;
    if(dataid>=n) return;
    tmp[dataid] =g_odata[dataid];  
    __syncthreads();
    for (int offset = 1; offset < gridDim.x ; offset *= 2)   
    {     
        if (dataid >= offset*blockDim.x)  
        {
            if((blockDim.x*blockIdx.x-((offset-1)*blockDim.x)-1)>=0){
                //all in all elements
                tmp[dataid] += tmp[blockDim.x*blockIdx.x-((offset-1)*blockDim.x)-1]; 
            }
             
        }
        __syncthreads();   
    }   
    g_odata[dataid] = tmp[dataid];
}*/


 __global__ void block_sum(int *d_v_r,int *tmp, int n) 
 {   
    int dataid = blockDim.x*blockIdx.x+threadIdx.x;
    if(blockIdx.x>0 && dataid<n)
       d_v_r[dataid]+=tmp[blockIdx.x-1];
}
 
int main() {
    // Vector size
    int N = 1 << 12;
    //N = 50;
    size_t bytes = N * sizeof(int);
 
    // Host data
    vector<int> h_v(N);
    vector<int> h_v_r(N);
    
 
    // Initialize the input data
  
    init(h_v);
     /*for(int i=0;i<N;++i){
        cout<<h_v[i]<<" ";
    }
    cout<<"\n";*/
    // Allocate device memory
    int *d_v, *d_v_r,*tmp,*tmp2;
    hipMallocManaged(&d_v, bytes);
    hipMallocManaged(&d_v_r, bytes);
    
    
    // Copy to device
    hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);
    
    /*for(int i=0;i<N;++i){
        cout<<d_v[i]<<" ";
    }*/
    // TB Size
    cout<<"\n";
    const int TB_SIZE = 32;
 
    // Grid Size (No padding)
    int GRID_SIZE = (N+TB_SIZE-1)/TB_SIZE;
     hipMallocManaged(&tmp, GRID_SIZE*sizeof(int));
      hipMallocManaged(&tmp2, GRID_SIZE*sizeof(int));
    // Call kernels
      
    scan<<<GRID_SIZE,TB_SIZE>>>(d_v_r,d_v,tmp,N);
    scan<<<1,GRID_SIZE>>>(tmp2,tmp,tmp,GRID_SIZE);
    //block_scan<<<GRID_SIZE,TB_SIZE>>>(d_v_r,tmp,N);
    block_sum<<<GRID_SIZE,TB_SIZE>>>(d_v_r,tmp2,N);
   
 
    // Copy to host;
    hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);
    /* for(int i=0;i<N;++i){
        cout<<h_v_r[i]<<" ";
    }*/
    int t=0;
    for(int i=0;i<N;i++){
        t+=h_v[i];
        if(t!=h_v_r[i]){
            cout<<"Failed";
            cout<<i;
            break;
        }
    }
    cout << "COMPLETED SUCCESSFULLY\n";
 
    return 0;
}